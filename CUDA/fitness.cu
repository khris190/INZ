#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cuPrintf.cu"
#include "../include/Util/Profiler.hpp"
__global__ void fitness_v1_RGBA2(int n, int width, unsigned char *pA, unsigned char *pB, float * ret)
{   
    int i  = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<n)
    {
        float absR = fabsf(pA[i*4 + 0] - pB[i*4 + 0]);
        float absG = fabsf(pA[i*4 + 1] - pB[i*4 + 1]);
        float absB = fabsf(pA[i*4 + 2] - pB[i*4 + 2]);
        float absA = fabsf(pA[i*4 + 3] - pB[i*4 + 3]);
        float val2 = (absR + absG + absB) + absA;
        ret[i] = (float)(255.f - val2 / 4.f) / 255.f;
        
    }
}


float calculateFitness(unsigned char *img_data, unsigned char *surface_data, int _width, int _height){

    unsigned char *x, *y;
    float *ret;
    // Allocate Unified Memory – accessible from CPU or GPU
    int size = _width * _height;


    hipMallocManaged(&x, 4 * size *sizeof(unsigned char));
    hipMallocManaged(&y, 4 * size *sizeof(unsigned char));
    hipMallocManaged(&ret, size *sizeof(float));
    
    
    hipMemcpy(x, img_data, 4 * _width * _height, hipMemcpyDefault);
    hipMemcpy(y, surface_data, 4 * _width * _height, hipMemcpyDefault);


    fitness_v1_RGBA2 <<<_width,_height>>>(size, _width, x, y, ret);
    hipError_t ce = hipGetLastError();
    hipDeviceSynchronize();
    float result = 0;
    int temp_offset;
    float row_fitness = 0;
    {
        newTimer("sumUpFitness");
        for (size_t y = 0; y < _height; y++)
        {
            for (size_t x = 0; x < _width; x++)
            {
                temp_offset = y * _width + x;
                row_fitness += ret[temp_offset];
            }
            row_fitness /= _width;
            result += row_fitness;
            row_fitness = 0;
        }
        result /= _height;
    }

    // Free memory
    hipFree(x);
    hipFree(y);
    hipFree(ret);
    return (result);
}